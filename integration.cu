#include "hip/hip_runtime.h"
#include<stdio.h>
#include<nbody.h>

__global__
void velocities(void *A, void *V){
  //input: pointers to acceleration and velocity
  //output: update velocity
        float3 *a = (float3 *)A;
        float3 *v = (float3 *)V;
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        v[id].x += dt*a[id].x;
        v[id].y += dt*a[id].y;
        v[id].z += dt*a[id].z;
}

__global__
void positions(void *V, void *P){
  //input: pointers to velocity and position
  //output: update position
        float4 *pos = (float4 *)P;
        float3 *v = (float3 *)V;
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        pos[id].x += dt*v[id].x;
        pos[id].y += dt*v[id].y;
        pos[id].z += dt*v[id].z;
        printf("%d's position is (%f, %f, %f)\n", id, pos[id].x, pos[id].y, pos[id].z);

}

