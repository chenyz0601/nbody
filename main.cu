#include "hip/hip_runtime.h"
#include <stdio.h>
#include <nbody.h>

int main(){

        int size4 = sizeof(float4)*N;
        int size3 = sizeof(float3)*N;
        float4 *c_bodies;
        hipMalloc((void**)&c_bodies,size4);
        float3 *c_vel;
        hipMalloc((void**)&c_vel,size3);
        float3 *c_a;
        float4 bodies[N];
        float3 vel[N], a[N] = {0.0f};
        initialization(bodies, vel, N);
        //allocate variables in cuda memory
        hipMalloc((void**)&c_a,size3);
        hipMemcpy( c_bodies, bodies, size4, hipMemcpyHostToDevice );
        hipMemcpy( c_vel, vel, size3, hipMemcpyHostToDevice );
        hipMemcpy( c_a, a, size3, hipMemcpyHostToDevice );
        //do the time integration
        for (int i = 0; i < MAX; i ++){
                printf("frame is: %d\n", i);
                accelerations<<<BLOCK, BLOCKDIM, sizeof(float4)*BLOCKDIM>>>(c_bodies, c_a);
                velocities<<<BLOCK, BLOCKDIM>>>(c_a, c_vel);
                positions<<<BLOCK, BLOCKDIM>>>(c_vel, c_bodies);
        }
        //copy the results from cuda memory to cup memory
        hipMemcpy( bodies, c_bodies, size4, hipMemcpyDeviceToHost );
        hipMemcpy( vel, c_vel, size3, hipMemcpyDeviceToHost );
        hipMemcpy( a, c_a, size3, hipMemcpyDeviceToHost );
        for (int i = 0; i < N; i ++){
          printf("ax is %f\n", a[i].x);
        }
        hipFree(c_bodies);
        hipFree(c_vel);
        hipFree(c_a);  
        return EXIT_SUCCESS;
}

