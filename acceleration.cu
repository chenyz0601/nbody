#include "hip/hip_runtime.h"
#include<stdio.h>
#include<nbody.h>

__device__
float3 acc(float4 pi, float4 pj, float3 ai){
   // input: positions of i-th an j-th body and the current acceleration of i-th body 
   // output: acceleration of i-th body
        float3 r;
        r.x = pj.x - pi.x;
        r.y = pj.y - pi.y;
        r.z = pj.z - pi.z;
        float distSqr = r.x*r.x+r.y*r.y+r.z*r.z+eps2;
        float dist = pow(distSqr, -1.5f);
        float temp = G*dist*pj.w;
        ai.x += temp*r.x;
        ai.y += temp*r.y;
        ai.z += temp*r.z;
        return ai;
}

__device__
float3 tile(float4 privateP, float3 a){
  //input: a thread's own position and its acceleration
  //output: acceleration
        int i;
        extern __shared__ float4 sharedP[];
        for (i = 0; i < blockDim.x; i ++) {
                a = acc(privateP, sharedP[i], a);
        }
        return a;
}

__global__
void accelerations(void *X, void *A){
  //input: pointers to position and acceleration in global memory
  //output: acceleration 
        extern __shared__ float4 sharedP[];//prepare shared datas
        float4 *globalX = (float4 *)X;
        float3 *globalA = (float3 *)A;
        float4 privateP;
        int i, t;//i the index of bodies; t the index of tiles
        float3 a = {0.0f,0.0f,0.0f};
        int id = blockIdx.x*blockDim.x+threadIdx.x;//id the index of thread
        privateP = globalX[id];
        for (i = 0, t = 0; i < N; i += BLOCKDIM, t ++){
                int idx = t*blockDim.x+threadIdx.x;
                sharedP[threadIdx.x]  = globalX[idx];
                __syncthreads();//joint all threads in one block after initializing shared datas 
                a = tile(privateP, a);
                __syncthreads();//joint all threads in one block before entering next tile
        }
        globalA[id]  = a;//assign acceleration to the global memory
        printf("%d's acc is (%f, %f, %f)\n", id, a.x, a.y, a.z);
}
