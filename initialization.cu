#include "hip/hip_runtime.h"
#include<stdio.h>
#include<nbody.h>

void initialization(float4 *P, float3 *V, int num){
        float4 *pos = (float4 *)P;
        float3 *v = (float3 *)V;
        int id;
        for (id = 0; id < num; id ++){
          pos[id].w = 1.0;//POS.w;
          pos[id].x = 1.0*id;//POS.x*id;
          pos[id].y = 1.0*id;//POS.y*id;
          pos[id].z = 1.0*id;//POS.z*id;
          v[id].x = 1.0;//VEL.x;
          v[id].y = 1.0;//VEL.y;
          v[id].z = 1.0;//VEL.z;
          printf("%d's initialize: pos = (%f, %f, %f,) vel = (%f, %f, %f)\n", id, pos[id].x, pos[id].y, pos[id].z, v[id].x, v[id].y, v[id].z);
        }
}

